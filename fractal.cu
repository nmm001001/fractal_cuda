#include "hip/hip_runtime.h"
/*
Computing a movie of zooming into a fractal

Original C++ code by Martin Burtscher, Texas State University

Reference: E. Ayguade et al., 
           "Peachy Parallel Assignments (EduHPC 2018)".
           2018 IEEE/ACM Workshop on Education for High-Performance Computing (EduHPC), pp. 78-85,
           doi: 10.1109/EduHPC.2018.00012

*/

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "timer.h"
#include "fractal.h"

static const double Delta = 0.001;
static const double xMid =  0.23701;
static const double yMid =  0.521;

__global__ void compute_frames(double aspect_ratio, int width, int height, int num_frames, unsigned char *picture_array) {
  int row = blockIdx.y*blockDim.y+threadIdx.y;
  int col = blockIdx.x*blockDim.x+threadIdx.x;
  int frame_index = blockIdx.z;
  if (frame_index >= num_frames) {
    return;
  };

  double delta = Delta * pow(0.98, frame_index);

  const double x0 = xMid - delta * aspect_ratio;
  const double y0 = yMid - delta;
  const double dx = 2.0 * delta * aspect_ratio / width;
  const double dy = 2.0 * delta / height;
  

  if (col <= width && row <= height) {
    
    const double cy = fma(dy, row, y0);
    const double cx = fma(dx, col, x0);

    double x = cx;
    double y = cy;
    int depth = 256;

    double x2;
    double y2;

    do {
      x2 = x*x;
      y2 = y*y;
      y = 2*x*y+cy;
      x = x2-y2+cx;
      depth--;
    } while ((depth > 0) && ((x2+y2) < 5.0));
    picture_array[frame_index * height * width + row * width + col] = (unsigned char) depth;
  }
}

int main(int argc, char *argv[]) {
  float start, end;

  printf("Fractal v1.6 [parallel]\n");

  /* read command line arguments */
  if (argc != 4) {fprintf(stderr, "usage: %s height width num_frames\n", argv[0]); exit(-1);}
  int width = atoi(argv[1]);
  if (width < 10) {fprintf(stderr, "error: width must be at least 10\n"); exit(-1);}
  int height = atoi(argv[2]); 
  if (height < 10) {fprintf(stderr, "error: height must be at least 10\n"); exit(-1);}
  int num_frames = atoi(argv[3]);
  if (num_frames < 1) {fprintf(stderr, "error: num_frames must be at least 1\n"); exit(-1);}
  printf("Computing %d frames of %d by %d fractal\n", num_frames, width, height);

  
  /* allocate image array */
  int pic_size = (sizeof(unsigned char) * num_frames * height * width);
  unsigned char *device_picture;
  unsigned char *host_picture = (unsigned char *)malloc(pic_size);
  GET_TIME(start);
  hipMalloc(&device_picture, pic_size);
  
  
  dim3 threads_per_block(32, 32, 1);
  dim3 num_blocks((width+threads_per_block.x-1)/threads_per_block.x, 
  (height+threads_per_block.y-1)/threads_per_block.y, num_frames);

  double aspect_ratio = (double)width / (double)height;
  
  
  compute_frames<<<num_blocks, threads_per_block>>>(aspect_ratio, width, height, num_frames, device_picture);
  

  hipError_t cuda_err = hipGetLastError();
  if (cuda_err != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cuda_err));
  }
  hipDeviceSynchronize();
  hipMemcpy(host_picture, device_picture, pic_size, hipMemcpyDeviceToHost);
  hipFree(device_picture);
  GET_TIME(end);

  /* end time */
 
  float elapsed = end - start;
  printf("Parallel compute time: %.4f s\n", elapsed);

  /* write frames to BMP files */
  if ((width <= 320) && (num_frames <= 100)) { /* do not write if images large or many */
    for (int frame = 0; frame < num_frames; frame++) {
      char name[32];
      sprintf(name, "fractal%d.bmp", frame + 1000);
      writeBMP(width, height, &host_picture[frame * height * width], name);
    }
  }


  free(host_picture);


  return 0;
} /* main */

